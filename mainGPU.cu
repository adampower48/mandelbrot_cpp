#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <iostream>
#include <fstream>
#include <thrust/complex.h>

__device__
void mandelbrot(double x, double y, double bound, int maxIters, int *outArr, int idx) {
    using namespace std::complex_literals;
    using namespace thrust;

    // Check for value inside bulbs
    double p = (x - 0.25) * (x - 0.25) + y * y;
    if (p * (p + (x - 0.25)) <= 0.25 * y * y) {
        outArr[idx] = 0;
        return;
    }

    complex<double> c = complex<double>(x, y);
    complex<double> z;

    int i = 0;
    while (i < maxIters & norm(z) < bound * bound) {
        z = z * z + c;
        i++;
    }

    if (i == maxIters) {
        outArr[idx] = 0;
    } else {
        outArr[idx] = i;
    }
}

__global__
void mandelbrotRow(double y, double xmin, double stepSize, int numSteps, double bound, int maxIters, int *outArr,
                   int idxStart) {
    for (int j = 0; j < numSteps; ++j) {
        double x = xmin + stepSize * j;
        mandelbrot(x, y, bound, maxIters, outArr, idxStart + j);
    }
}

__global__
void
mandelbrotSingle(double xmin, double ymin, double stepSize, int numSteps, double bound, int maxIters, int *outArr) {
    int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    int idxY = blockIdx.y * blockDim.y + threadIdx.y;
    if (idxX >= numSteps || idxY >= numSteps)
        return;

    double x = xmin + stepSize * idxX;
    double y = ymin + stepSize * idxY;
    int idx = numSteps * idxY + idxX;
    mandelbrot(x, y, bound, maxIters, outArr, idx);
}


void mandelbrotSquareArr(double centreX, double centreY, double size, int numSteps, double bound, int maxIters,
                         int *outArr) {
    double stepSize = size / numSteps;
    double xmin = centreX - size / 2;
    double ymin = centreY - size / 2;
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(std::ceil((double) numSteps / threadsPerBlock.x),
                   std::ceil((double) numSteps / threadsPerBlock.y));


    mandelbrotSingle << < numBlocks, threadsPerBlock >> > (xmin, ymin, stepSize, numSteps, bound, maxIters, outArr);
    hipDeviceSynchronize();

}

void writeArrToFile(int width, int height, int *outArr, const std::string &filename) {
    std::ofstream file;
    file.open(filename);

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            file << outArr[i * height + j];
            if (j < width - 1)
                file << ",";
        }
        file << std::endl;
    }

    file.close();
}

void writeArrToCout(int width, int height, int *outArr) {
    using namespace std;

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            cout << outArr[i * height + j];
            if (j < width - 1)
                cout << ",";
        }
        cout << endl;
    }

}


int main() {
    // Generate mandelbrot results
    int n = 100;
//    int *mandels = new int[n * n];
    int *mandels;
    hipMallocManaged(&mandels, n * n * sizeof(int));
//    mandelbrotSquareArr(0.02445758009307159, 0.6296933276014198, 0.000001, n, 2, 10000, mandels);
    mandelbrotSquareArr(-1.5636314569242658, 0.000016435977612209248, 0.00000000001, n, 2, 10000, mandels);
//    mandelbrotSquareArr(-0.5, 0, 2, n, 2, 250, mandels);

//    writeArrToFile(n, n, mandels, "mandelbrot.csv");
    writeArrToCout(n, n, mandels);


    hipFree(mandels);

    return 0;
}